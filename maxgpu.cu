#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


// Thread block size
#define BLOCK_SIZE 512

int SOA;



int i;
void randomInit(int* data, int size)
{	
    for( i = 0; i < size; i++){
       data[i] = rand()  % size;   
   }
}
 

__global__ void ReductionMax2(int *input, int *results, int n)    //take thread divergence into account
{	
	extern __shared__ int sdata[];
	//sdata[blockIdx.x] =0;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; // thread Id for particular block
	unsigned int tx = threadIdx.x;// thread Id for current block
	 //load input into __shared__ memory 
	int x; 
	int j;
	//printf("%d " , i);
	__syncthreads();

for(x=blockIdx.x*512; x<(blockIdx.x*512)+512; x++){
if(x < n && tx==0){ 
	//printf("blockIdx = %d and x = %d\n", blockIdx.x,x);
if(input[x]> results[blockIdx.x]){
	results[blockIdx.x] = input[x];
	//printf("results = %d", results[blockIdx.x]);
	}
	}
}
__syncthreads();
if(threadIdx.x==0 && blockIdx.x==0){
for(j=0; j<(blockDim.x/512)+1; j++){
//printf("results[j] = %d\n", results[j]);
}
}


}// end of kernel




int main(int argc, char* argv[]) 
{ 

int counter =0;
int gpuMax=0;
int cpuMax=0;

SOA = atol(argv[1]);


if( SOA >= 10000000){
counter = SOA/1000000;
}

if( counter==0){
		int num_blocks = SOA / BLOCK_SIZE;
		//allocate host memory for array a
		unsigned int mem_size_a = sizeof(int) * SOA;
		int* h_a = (int*)malloc(mem_size_a);
		
		//initialize host memory
		randomInit(h_a,SOA);

		//allocate device memory
		int* d_a;
		hipMalloc((void**) &d_a, mem_size_a);

		//copy host memory to device
		hipMemcpy(d_a, h_a, mem_size_a, hipMemcpyHostToDevice);


		unsigned int mem_size_b = sizeof(int) * SOA;
		int* d_b;

		hipMalloc((void**) &d_b, mem_size_b);
	

	
	 int mem_size_c = sizeof(int) * SOA;
		int* h_c = (int*)malloc(mem_size_c);

		//setup execution parameters
		dim3 block(512);
		dim3 grid(num_blocks+1);

		//execute the kernel
		ReductionMax2<<<grid, block>>>(d_a,d_b,SOA);
		hipMemcpy(h_c, d_b, mem_size_c, hipMemcpyDeviceToHost);

		int i;
	for(i=0; i<num_blocks+1;i++){
		if(h_c[i] > gpuMax){
		gpuMax = h_c[i];
		}
		}
	
 printf(" The maximum number in the array is: %d\n", gpuMax);

		//clean up memory
		free(h_a);
		free(h_c);
		hipFree(d_a);
		hipFree(d_b);

		hipDeviceReset();
}// end of if
else{
int gpuMax=0;
int tempMax=0;
unsigned int mem_size_a = sizeof(int) * SOA;
int* h_a = (int*)malloc(mem_size_a);
//initialize host memory
randomInit(h_a,SOA);
int i;
int j;
SOA = SOA/counter;
int num_blocks = SOA / BLOCK_SIZE;
printf("SOA is %d\n", SOA);
for (i=0; i<counter; i++){
int * h_b = (int*) malloc(sizeof(int)*SOA);
for(j= i*SOA; j<i*SOA+SOA; j++){
	h_b[j] = h_a[j];
}// end of initiating random array 
	int* d_a;
	hipMalloc((void**) &d_a, mem_size_a);
	hipMemcpy(d_a, h_b, (sizeof(int)*SOA), hipMemcpyHostToDevice);
	
	int* d_b;
	hipMalloc((void**) &d_b, mem_size_a);

	 int mem_size_c = sizeof(int) * SOA;
	int* h_c = (int*)malloc(mem_size_c);
		dim3 block(512);
		dim3 grid(num_blocks);

		//execute the kernel
		//first reduce per-block partial maxs
		ReductionMax2<<<grid, block>>>(d_a,d_b,SOA);
		hipMemcpy(h_c, d_b, mem_size_c, hipMemcpyDeviceToHost);
		for(i=0; i<num_blocks+1;i++){
		if(h_c[i] > tempMax){
		tempMax = h_c[i];
		}
		}
if (tempMax>gpuMax){
gpuMax = tempMax;
}
} //end of for loop
 printf(" The maximum number in the array is: %d\n", gpuMax);
}// end else
}// end of main